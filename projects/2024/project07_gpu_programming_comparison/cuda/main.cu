#include "device.cuh"
#include "host.cuh"

#include <chrono>
#include <fstream>
#include <sstream>

#ifndef CUDART_VERSION
#error "CUDART_VERSION not defined!"
#endif


using time_point = std::chrono::time_point<std::chrono::steady_clock>;


// run_simulation<T>():
// Runs the 4th-order diffusion simulation on a CUDA-enabled GPU and writes its results to some output files.
//
// Input:   xsize, ysize, zsize :: Dimensions of the domain (including boundary points)
//          itrs                :: Number of timestep iterations
//          bdry                :: Number of boundary points
//          mode                :: Computation mode (double-5/13-point stencil(s) with/without shared memory)
//          T                   :: Numeric real type
// Output:  return (...)        :: Measured time (memory transfer + device allocation + computation) in seconds
template<typename T>
double run_simulation(std::size_t xsize, std::size_t ysize, std::size_t zsize, std::size_t itrs, std::size_t bdry, Mode mode) {

    constexpr T alpha = static_cast<T>(1) / 32;
    const std::size_t xmin = bdry, xmax = xsize - bdry;
    const std::size_t ymin = bdry, ymax = ysize - bdry;

    hipStream_t stream;
    T *u, *v, *u_host;
    std::ofstream os;

    check(hipHostMalloc(&u_host, xsize * ysize * zsize * sizeof(T)));
    host::initialise(u_host, xsize, ysize, zsize);

    os.open("in_field.csv");
    host::write_file(os, u_host, xsize, ysize, zsize);
    os.close();

    const time_point begin = std::chrono::steady_clock::now();
    check(hipStreamCreate(&stream));

    #if CUDART_VERSION >= 11020
    // Requires at least CUDA v11.2 to be installed.
    check(hipMallocAsync(&u, xsize * ysize * zsize * sizeof(T), stream));
    check(hipMallocAsync(&v, xsize * ysize * zsize * sizeof(T), stream));
    #else
    check(hipMalloc(&u, xsize * ysize * zsize * sizeof(T)));
    check(hipMalloc(&v, xsize * ysize * zsize * sizeof(T)));
    #endif

    check(hipMemcpyAsync(u, u_host, xsize * ysize * zsize * sizeof(T), hipMemcpyHostToDevice, stream));

    switch(mode) {
        case Mode::laplap_global: {
            for(std::size_t i = 0; i < itrs; ++i) {
                device::update_boundaries(stream, u, xmin, xmax, ymin, ymax, xsize, ysize, zsize);
                device::update_interior_double_laplacian(stream, u, v, alpha, xmin, xmax, ymin, ymax, xsize, ysize, zsize);
            }
            break;
        }
        case Mode::laplap_shared: {
            for(std::size_t i = 0; i < itrs; ++i) {
                device::update_boundaries(stream, u, xmin, xmax, ymin, ymax, xsize, ysize, zsize);
                device::update_interior_double_laplacian_shared(stream, u, v, alpha, xmin, xmax, ymin, ymax, xsize, ysize, zsize);
            }
            break;
        }
        case Mode::biharm_global: {
            for(std::size_t i = 0; i < itrs; ++i) {
                device::update_boundaries(stream, u, xmin, xmax, ymin, ymax, xsize, ysize, zsize);
                device::update_interior_biharmonic(stream, u, v, alpha, xmin, xmax, ymin, ymax, xsize, ysize, zsize);
            }
            break;
        }
        case Mode::biharm_shared: {
            for(std::size_t i = 0; i < itrs; ++i) {
                device::update_boundaries(stream, u, xmin, xmax, ymin, ymax, xsize, ysize, zsize);
                device::update_interior_biharmonic_shared(stream, u, v, alpha, xmin, xmax, ymin, ymax, xsize, ysize, zsize);
            }
            break;
        }
        default: __builtin_unreachable();
    }
    device::update_boundaries(stream, u, xmin, xmax, ymin, ymax, xsize, ysize, zsize);

    check(hipMemcpyAsync(u_host, u, xsize * ysize * zsize * sizeof(T), hipMemcpyDeviceToHost, stream));

    #if CUDART_VERSION >= 11020
    // Requires at least CUDA v11.2 to be installed.
    check(hipFreeAsync(u, stream));
    check(hipFreeAsync(v, stream));
    #else
    check(hipFree(u));
    check(hipFree(v));
    #endif

    check(hipStreamDestroy(stream));
    check(hipDeviceSynchronize());
    const time_point end = std::chrono::steady_clock::now();

    os.open("out_field.csv");
    host::write_file(os, u_host, xsize, ysize, zsize);
    os.close();

    check(hipHostFree(u_host));

    return std::chrono::duration<double, std::milli>(end - begin).count() / 1000;
}


// templated_main<T>():
// Main function with flexible numeric real type.
//
// Input:   argv            :: Input arguments
//          argc            :: Number of input arguments
//          T               :: Numeric real type
// Output:  return (...)    :: Exit code (EXIT_SUCCESS or EXIT_FAILURE)
template<typename T>
int templated_main(int argc, char const **argv) {

    if(argc == 7) {
        std::size_t nx, ny, nz, bdry, itrs;
        Mode mode;

        {
            std::istringstream nx_ss(argv[1]), ny_ss(argv[2]), nz_ss(argv[3]), bdry_ss(argv[4]), itrs_ss(argv[5]);
            nx_ss >> nx; ny_ss >> ny; nz_ss >> nz; bdry_ss >> bdry; itrs_ss >> itrs;
            mode = utils::mode_from_string(argv[6]);

            if(nx_ss.fail() || ny_ss.fail() || nz_ss.fail() || itrs_ss.fail() ||
               nx == 0 || ny == 0 || nz == 0 || bdry < 2 || itrs == 0 || mode == Mode::invalid) {

                utils::print_args_errmsg();
                return EXIT_FAILURE;
            }
        }

        std::cout << "================================================================================\n";
        std::cout << "                             Welcome to stencil2d!\n";
        std::cout << "Version    :: C++ with CUDA v" << CUDART_VERSION / 1000 << '.' << CUDART_VERSION / 10 % 100 << '\n';
        std::cout << "Interior   :: (" << nx << ", " << ny << ", " << nz << ")\n";
        std::cout << "Boundaries :: (" << bdry << ", " << bdry << ", " << 0 << ")\n";
        std::cout << "Iterations :: " << itrs << '\n';
        std::cout << "Real size  :: " << sizeof(T) << '\n';
        std::cout << "Exec. mode :: " << utils::get_mode_desc(mode) << '\n';
        std::cout << "================================================================================\n";

        const double time = run_simulation<T>(nx + 2 * bdry, ny + 2 * bdry, nz, itrs, bdry, mode);

        std::cout << "Runtime    :: " << std::fixed << std::setprecision(10) << time << "s\n";
        std::cout << "================================================================================\n";
    }
    else {
        utils::print_args_errmsg();
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}


int main(int argc, char const **argv) {
    #if !defined(REALSIZE) || REALSIZE == 8
    return templated_main<double>(argc, argv);
    #elif REALSIZE == 4
    return templated_main<float>(argc, argv);
    #else
    #error "Selected REALSIZE not supported!"
    #endif
}
